#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include "pgmUtility.h"
#include "pgmProcess.h"

int *pgmRead(char **header, int *numRows, int *numCols, FILE *in){
    int i, j;
    
    // read in header of the image first
    for( i = 0; i < rowsInHeader; i ++)
    {
        if ( header[i] == NULL )
        {
            return NULL;
        }
        if( fgets( header[i], maxSizeHeadRow, in ) == NULL )
        {
            return NULL;
        }
    }

    // extract rows of pixels and columns of pixels
    puts(header[rowsInHeader - 2]);
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows );  // in pgm the first number is # of cols
  
    int *pixels = (int*)malloc((*numCols) * (*numRows) * sizeof(int));

    
    // read in all pixels into the pixels array.

    for( i = 0; i < *numRows; i ++ )
        for( j = 0; j < *numCols; j ++ )
            if(fscanf(in, "%d ", &pixels[(i*(*numCols))+j]) < 0 )
                return NULL;

    
    return pixels;
}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ){
    
    int num_bytes = (numCols*numRows*sizeof(int));
    int *d_a;
    hipMalloc((void**)&d_a, num_bytes);
    hipMemcpy(d_a, pixels, num_bytes, hipMemcpyHostToDevice);

    dim3 grid, block;
    block.x = 16;
    block.y = 16;
    grid.x = ceil((float)numCols/block.x);
    grid.y = ceil((float)numRows/block.y);
    
    clock_t before = clock();
    pgmDrawCircleK<<<grid, block>>>(d_a, numRows, numCols, centerRow, centerCol, radius);
    clock_t difference = clock() - before;
    double msec = (double)difference * 1000000 / CLOCKS_PER_SEC;
    printf("%s%f%s\n","Kernel run time: ", msec, " microseconds");
    
    hipMemcpy(pixels, d_a, num_bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);

    return 0;
}

int pgmDrawEdge( int *pixels, int numRows, int numCols, int edgeWidth, char **header ){
    int num_bytes = (numCols*numRows*sizeof(int));
    int *d_a;

    hipMalloc((void**)&d_a, num_bytes);
    hipMemcpy(d_a, pixels, num_bytes, hipMemcpyHostToDevice);

    dim3 grid, block;
    block.x = 16;
    block.y = 16;
    grid.x = ceil((float)numCols/block.x);
    grid.y = ceil((float)numRows/block.y);
    
    clock_t before = clock();
    pgmDrawEdgeK<<<grid,block>>>(d_a, numRows, numCols, edgeWidth);
    clock_t difference = clock() - before;
    double msec = (double)difference * 1000000 / CLOCKS_PER_SEC;
    printf("%s%f%s\n","Kernel run time: ", msec, " microseconds");
    
    hipMemcpy(pixels, d_a, num_bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);

    return 0;
}

int pgmDrawLine( int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ){
    int num_bytes = (numCols*numRows*sizeof(int));
    int *d_a;

    hipMalloc((void**)&d_a, num_bytes);
    
    if(0==d_a)
    {
    	printf("Couldn't allocate memory for d_a\n");
    	return 1;
    }
    
    hipMemcpy(d_a, pixels, num_bytes, hipMemcpyHostToDevice);

    dim3 grid, block;
    block.x = 16;
    block.y = 16;
    grid.x = ceil((float)numCols/block.x);
    grid.y = ceil((float)numRows/block.y);
    
    clock_t before = clock();
    pgmDrawLineK<<<grid, block>>>(d_a, numRows, numCols, p1row, p1col, p2row, p2col);
    clock_t difference = clock() - before;
    double msec = (double)difference * 1000000 / CLOCKS_PER_SEC;
    printf("%s%f%s\n","Kernel run time: ", msec, " microseconds");
    
    hipMemcpy(pixels, d_a, num_bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);

    return 0;
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out )
{
    int i, j;
    // write the header
    for ( i = 0; i < rowsInHeader; i ++ )
    {
        fprintf(out, "%s", *( header + i ) );
    }
    
    // write the pixels
    for( i = 0; i < numRows; i ++ )
    {
        for ( j = 0; j < numCols; j ++ )
        {
            if ( j < numCols - 1 )
                fprintf(out, "%d ", pixels[(i*numCols)+j]);
            else
                fprintf(out, "%d\n", pixels[(i*numCols)+j]);
        }
    }
    return 0;
}

